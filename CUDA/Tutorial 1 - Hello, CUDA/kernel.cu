#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

__global__ void simple_add(const int *A, const int *B, int *C)
{
	C[threadIdx.x] = A[threadIdx.x] + B[threadIdx.x];
}

int main(int arg, char* args[])
{
	const int size = 10;
	int A[] = { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9 };
	int B[] = { 0, 1, 2, 0, 1, 2, 0, 1, 2, 0 };
	int C[size];

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
	{
		std::cout << "No CUDA devices found!" << std::endl;
		exit(1);
    }

	int *buffer_A = 0;
    int *buffer_B = 0;
    int *buffer_C = 0;
    hipError_t cudaStatus;

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

	std::cout << "Using device: " << prop.name << std::endl;
	
    // Allocate GPU buffers for three vectors (two input, one output).
	hipMalloc((void**)&buffer_A, size * sizeof(int));
	hipMalloc((void**)&buffer_B, size * sizeof(int));
	hipMalloc((void**)&buffer_C, size * sizeof(int));
	
    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(buffer_A, A, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(buffer_B, B, size * sizeof(int), hipMemcpyHostToDevice);

	// Launch a kernel on the GPU with one thread for each element.
    simple_add<<<1, size>>>(buffer_A, buffer_B, buffer_C);

	// Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
	{
		std::cout << "Kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
		hipFree(buffer_A);
		hipFree(buffer_B);
		hipFree(buffer_C);
		exit(1);
    }
	
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
	{
		std::cout << "Could not synchronize device!" << std::endl;
		hipFree(buffer_A);
		hipFree(buffer_B);
		hipFree(buffer_C);
		exit(1);
    }
	
    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(C, buffer_C, size * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(buffer_A);
	hipFree(buffer_B);
	hipFree(buffer_C);

	if(cudaStatus != hipSuccess)
	{
		std::cout << "Could not copy buffer memory to host!" << std::endl;
		exit(1);
	}

    //Prints the array
	std::cout << "Result:" << std::endl;
	for (int i = 0; i < size; i++)
	{
		std::cout << A[i] << " + " << B[i] << " = " << C[i] << std::endl;
	}

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
	{
		std::cout << "Device reset failed!" << std::endl;
        exit(1);
    }

    return 0;
}