#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#include "PNG.h"

__global__ void boxFilter(const unsigned char* in, unsigned char* out, const int imageWidth, const int imageHeight, const int halfBoxWidth, const int halfBoxHeight)
{
	int x = blockIdx.x;
	int y = blockIdx.y;

	int count = 0;

	int index = (x + y * imageWidth) * 4;

	unsigned int total[4] = { 0, 0, 0, 0 };

	for (int i = -halfBoxWidth; i <= halfBoxWidth; i++)
	{
		for (int j = -halfBoxHeight; j <= halfBoxHeight; j++)
		{
			int cx = x + i;
			int cy = y + j;
			if (cx >= 0 && cy >= 0 && cx < imageWidth && cy < imageHeight)
			{
				int adjIndex = (cx + cy * imageWidth) * 4;
				for (int c = 0; c < 4; c++)
				{
					total[c] += static_cast<unsigned int>(in[adjIndex + c]);
				}
				count++;
			}
		}
	}

	out[index]     = static_cast<unsigned char>(total[0] / count);
	out[index + 1] = static_cast<unsigned char>(total[1] / count);
	out[index + 2] = static_cast<unsigned char>(total[2] / count);
	out[index + 3] = static_cast<unsigned char>(total[3] / count);
}

int main(int arg, char* args[])
{
	int filterWidth = 10;
	int filterHeight = 10;
	if (arg > 2)
	{
		filterWidth = std::atoi(args[1]);
		filterHeight = std::atoi(args[2]);
	}

	PNG inPng("Lenna.png");
	PNG outPng;
	outPng.Create(inPng.w, inPng.h);

	//store width and height so we can use them for our output image later
	const unsigned int w = inPng.w;
	const unsigned int h = inPng.h;
	//4 because there are 4 color channels R, G, B, and A
	int size = w * h * 4;

	unsigned char *in = 0;
	unsigned char *out = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		std::cout << "No CUDA devices found!" << std::endl;
		exit(1);
	}

	//prints the device the kernel will be running on
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	std::cout << "Using device: " << prop.name << std::endl;

	// Allocate GPU buffers for the images
	hipMalloc((void**)&in, size * sizeof(unsigned char));
	hipMalloc((void**)&out, size * sizeof(unsigned char));

	// Copy image data from host memory to GPU buffers.
	hipMemcpy(in, &inPng.data[0], size * sizeof(unsigned char), hipMemcpyHostToDevice);

	//free the input image because we do not need it anymore
	inPng.Free();

	// Launch a kernel on the GPU with one thread for each element.
	dim3 block_size(w, h);
	dim3 grid_size(1);
	boxFilter<<<block_size, 1>>>(in, out, w, h, filterWidth, filterHeight);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		std::cout << "Kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
		hipFree(in);
		hipFree(out);
		exit(1);
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		std::cout << "Could not synchronize device!" << std::endl;
		hipFree(in);
		hipFree(out);
		exit(1);
	}

	//temporary array to store the result from opencl
	auto tmp = new unsigned char[w * h * 4];
	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(tmp, out, size * sizeof(unsigned char), hipMemcpyDeviceToHost);
	hipFree(in);
	hipFree(out);

	//copy the data from the temp array to the png
	std::copy(&tmp[0], &tmp[w * h * 4], std::back_inserter(outPng.data));

	//write the image to file
	outPng.Save("cuda_tutorial_3.png");
	//free the iamge's resources since we are done with it
	outPng.Free();

	//free the temp array
	delete[] tmp;

	if (cudaStatus != hipSuccess)
	{
		std::cout << "Could not copy buffer memory to host!" << std::endl;
		exit(1);
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		std::cout << "Device reset failed!" << std::endl;
		exit(1);
	}

	return 0;
}